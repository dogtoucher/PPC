#include "hip/hip_runtime.h"
__global__ void mykernel(float* r, const float* d, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= n || j >= n)
        return;
    float v = HUGE_VALF;
    for (int k = 0; k < n; ++k) {
        float x = d[n*i + k];
        float y = d[n*k + j];
        float z = x + y;
        v = min(v, z);
    }
    r[n*i + j] = v;
}

void step(float* r, const float* d, int n) {
    // Allocate memory & copy data to GPU
    float* dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, n * n * sizeof(float)));
    float* rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(n, dimBlock.x), divup(n, dimBlock.y));
    mykernel<<<dimGrid, dimBlock>>>(rGPU, dGPU, n);
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}